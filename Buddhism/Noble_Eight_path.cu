
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float *d_input, float *d_output, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        d_output[tid] = d_input[tid] * d_input[tid];
    }
}

int main() {
    int size = 10;
    int numBytes = size * sizeof(float);
    
    // Input array on the host (CPU)
    float h_input[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    // Output array on the host (CPU)
    float h_output[10];

    // Allocate memory on the GPU
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, numBytes);
    hipMalloc((void**)&d_output, numBytes);

    // Copy data from host to device
    hipMemcpy(d_input, h_input, numBytes, hipMemcpyHostToDevice);

    // Define the number of threads per block and blocks per grid
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the CUDA kernel
    square<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

    // Copy the result from device to host
    hipMemcpy(h_output, d_output, numBytes, hipMemcpyDeviceToHost);

    // Print the results
    printf("Input: ");
    for (int i = 0; i < size; i++) {
        printf("%.2f ", h_input[i]);
    }
    printf("\n");

    printf("Output (Squared): ");
    for (int i = 0; i < size; i++) {
        printf("%.2f ", h_output[i]);
    }
    printf("\n");

    // Free allocated memory on the GPU
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
